//#include "utils.cuh"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <sys/time.h>
#include <string.h>
#include <math.h>
#include <limits.h>

#define BLOCKSIZE 1


struct MatrixStorage {
    int *row_ptr;
    int row_size;

    int *col_ind;
    int col_size;

    double *values;
    int val_size;
};

__host__ int* allocateArray(int size);
__host__ double* allocateDoubleArray(int size);

__host__ void initializeDoubleArray(double* arr, int size, double initalizer);
__host__ void initializeArray(int* arr, int size, int initalizer);
//void allocateMatrixStorage(struct MatrixStorage* matrixOut, int rows,int values);
__host__ void allocateMatrixStorage(int** row_ptr, int **col_ind,double **values, int row_size, int col_size, int val_size);
__host__  void initializeMatrixStorage(int **row_ptr, int **col_ind,double **values, int row_size, int col_size, int val_size);
__host__ void readMatrix(int **row_ptr, int **col_ind,double **values,
            int &row_size, int &col_size, int &val_size, char *filename);
__host__ void accumulateCounts(int* arr,int size,int val);
__host__ void printArray(int* arr, int size);
__host__ void printDoubleArray(double* arr, int size);
__host__ void matrix_vector_multip(int *row_ptr, int *col_ind, 
            double *values, int rows,int columns, double **x );

__host__ int* allocateArray(int size){
    int *arr;
    arr = (int *)malloc(sizeof(int) * size);
    return arr;
}
  
__host__ double* allocateDoubleArray(int size){
    double *arr;
    arr = (double *)malloc(sizeof(double) * size);
    return arr;
}

__host__ void initializeArray(int* arr, int size, int initalizer){
    int i;
    for (i = 0; i < size; i++){
        arr[i] = initalizer;
    }
}
__host__ void initializeDoubleArray(double* arr, int size, double initalizer){
    int i;
    for (i = 0; i < size; i++){
        arr[i] = initalizer;
    }
}

__host__ void allocateMatrixStorage(int** row_ptr, int **col_ind,double **values, int row_size,int col_size,int val_size){
    *row_ptr =  allocateArray(row_size);
    *col_ind =  allocateArray(val_size);
    *values =  allocateDoubleArray(val_size);
}

__host__ void initializeMatrixStorage(int** row_ptr, int **col_ind,double **values, int row_size,int col_size,int val_size){ 
    initializeArray(*row_ptr, row_size, 0);
    initializeArray(*col_ind, val_size, val_size);
    initializeDoubleArray(*values, val_size, (double) val_size);
}

__host__  void readMatrix(int **row_ptr, int **col_ind,double **values,
            int &row_size, int &col_size, int &val_size, char *filename){
   
    FILE * fileToRead;
    int  i;
    int row_index,col_index;
    double val;

    
    fileToRead = fopen(filename,"r");             //opens the file

    if (fileToRead == NULL){
         printf("file cant be found\n");
         return ;
    }else{
    
        fscanf(fileToRead,"%d %d %d", &(row_size), &(col_size), &(val_size));
        allocateMatrixStorage( row_ptr, col_ind, values, row_size,col_size,val_size);
        initializeMatrixStorage(row_ptr, col_ind, values,row_size,col_size, val_size);

        for (i = 0; i < val_size; i++)
        {   
            fscanf(fileToRead,"%d %d %lg", &row_index, &col_index, &val);
            //if (row_index == 5)
            //printf("%d,%d\n", row_ptr[row_index-1],row_index);
            (*row_ptr)[row_index-1] =  (*row_ptr)[row_index-1] + 1;	
            (*values)[i] = val;
            (*col_ind)[i] = col_index -1;
        } 
        //printf("\n");
    }
    fclose(fileToRead);
}

__host__ void accumulateCounts(int* arr,int size,int val){
    int i;
   
    if (arr[size-1] == 0){
        arr[size-1] = val;
    }
    else{
         arr[size-1] = val - arr[size-1];
    }
    
    for (i = size-2; i >=0; i--){
        if (arr[i] == 0)
            arr[i] = arr[i+1];
        else {
            arr[i] = arr[i+1] - arr[i];
        }
    }
}

__host__ void printArray(int* arr, int size){
     
    int i;
    for (i = 0; i < size; i++){
        printf("%d\n", arr[i]);
    }
    printf("\n");
}

__host__ void printDoubleArray(double* arr, int size){
    int i;
    for (i = 0; i < size; i++){
        printf("%lf\n", arr[i]);
    }
    printf("\n");
}

__host__ void matrix_vector_multip(int *row_ptr, int *col_ind, 
            double *values, int rows,int columns, double **x ){
    int i,j;
    double val = 0.0;
    
    double *newArr = allocateDoubleArray(rows);
    initializeDoubleArray(newArr,rows,0.0);
    
    double *tmpX = *x;

    for (i = 0; i < rows-1; i++){
        if (row_ptr[i] != row_ptr[i+1]){
            for (j = row_ptr[i]; j < row_ptr[i+1];j++){
                val = val + tmpX[col_ind[j]]*values[j];
            }
            newArr[i] = val;
        } else {
            newArr[i] = 0.0;
        }
        val = 0.0;
    }
    
    //printArray(col_ind,columns);
    i = rows -1;
    if (row_ptr[i] < rows){
        for (j = row_ptr[i]; j < columns;j++){
            
            val = val + tmpX[col_ind[j]]*values[j];
        }
        newArr[i] = val;
        
    }else {
        newArr[i] = 0.0;
    }
    val = 0.0;
    
    *x = newArr;
    newArr = NULL;
    free(tmpX);
}
__global__ void initializeDoubleArrayDev(double* arr, int size, double initalizer){
    int i;
    for (i = 0; i < size; i++){
        arr[i] = initalizer;
    }
}

__global__ void matrix_vector_multip_dev(int *row_ptr, int *col_ind, 
            double *values, int *rowsArr,int *columnsArr, double *x, double *newArr,int *single_block_size ){
    int block_size = single_block_size[0];
    int x_tid = blockIdx.x*blockDim.x + threadIdx.x;
    int y_tid = blockIdx.y*blockDim.y + threadIdx.y;
    int thread_index = x_tid + gridDim.x*block_size*y_tid;       
    

    int j;
    int rows = rowsArr[0];
    int columns = columnsArr[0];
    double val = 0.0;
    int ofset = rows * thread_index; 

    if (thread_index < rows){
        newArr[thread_index] = 0.0;
    }
    if (thread_index < rows -1 ){
        if (row_ptr[thread_index] != row_ptr[thread_index+1]){
            for (j = row_ptr[thread_index]; j < row_ptr[thread_index+1];j++){
                    val = val + x[col_ind[j]] *values[j];
            }
            newArr[thread_index] += val;
        }else {
            newArr[thread_index] += 0.0;
        }
    } else {
        if (thread_index == rows -1 && row_ptr[thread_index] < rows){
            for (j = row_ptr[thread_index]; j < columns;j++){
                val = val + x[col_ind[j]] *values[ofset + j];
            }
            newArr[thread_index] += val;
        }else {
            newArr[thread_index] += 0.0;
        }
    }

   
   __syncthreads();
    
}


int main(int argc, char** argv){

    int noThreads,number_of_iter,print_what;
    char *filename;

    if (argc < 5){
        printf("Please enter the all the following arguments:\n");
        printf("1. The number of threads used to compute Matrix-vector product\n");
        printf("2. The number of repetitions \n");
        printf("3. An argument to print on stdout \n");
        printf("4. Test-file name\n");
        exit(1);
    }else {
        noThreads = atoi(argv[1]);
        number_of_iter = atoi(argv[2]);
        print_what =  atoi(argv[3]);
        filename = argv[4];
    } 

    int *d_rowptr, *d_col_ind;
    double *d_values;
    int *d_rows, *d_columns;
    int *row_ptr, *col_ind, *single_block_size;
    double *values;
    int row_size, col_size, val_size;

    double * man_x, *man_x_result;
    double * d_man_x, *d_man_x_result;
 
    readMatrix(&row_ptr,&col_ind,&values,row_size,col_size,val_size ,filename);
    accumulateCounts(row_ptr,row_size,val_size);
    
    
    hipMalloc((void**)&d_rowptr, sizeof(int) * row_size);
    hipMalloc((void**)&d_col_ind, sizeof(int) * val_size);
    hipMalloc((void**)&d_values, sizeof(double) * val_size);
    hipMalloc((void**)&d_rows, sizeof(int));
    hipMalloc((void**)&d_columns, sizeof(int));
    hipMalloc((void**)&single_block_size, sizeof(int));
   
    hipMalloc((void**)&man_x, sizeof(double) * row_size);
    hipMalloc((void**)&man_x_result, sizeof(double) * row_size);
    hipMalloc((void**)&d_man_x, sizeof(double) * row_size);
    hipMalloc((void**)&d_man_x_result, sizeof(double) * row_size);


    man_x = allocateDoubleArray(row_size);
    man_x_result = allocateDoubleArray(row_size);
    initializeDoubleArray(man_x,row_size,1.0);
 
   
    hipMemcpy(d_rowptr, row_ptr, sizeof(int) * row_size, hipMemcpyHostToDevice);
    hipMemcpy(d_col_ind, col_ind, sizeof(int) * val_size, hipMemcpyHostToDevice);
    hipMemcpy(d_values, values, sizeof(double) * val_size, hipMemcpyHostToDevice);
    hipMemcpy(d_rows, &row_size, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_columns, &col_size, sizeof(int), hipMemcpyHostToDevice);
    
    int block_size = BLOCKSIZE;
    int a_block_can_hold =block_size*block_size;
    int blocks_in_a_grid;
    if (row_size % a_block_can_hold != 0){
        blocks_in_a_grid=row_size/a_block_can_hold+1;
    }else {
        blocks_in_a_grid=(row_size/a_block_can_hold+1)-1;
    }
    
    int grids = noThreads /(blocks_in_a_grid*a_block_can_hold);

    if (noThreads%(blocks_in_a_grid*a_block_can_hold)>0){
        grids++;
    }

    dim3 dimBlock(block_size,block_size);
    dim3 dimGrid(grids,blocks_in_a_grid);

    double *pt;
    hipMemcpy(single_block_size , &block_size, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_man_x, man_x, sizeof(double) * row_size, hipMemcpyHostToDevice);
    
    struct timeval t1, t2;

    gettimeofday(&t1, 0);
    for(int iter=0; iter<number_of_iter; iter++) {
        matrix_vector_multip_dev<<<dimGrid,dimBlock>>>(d_rowptr, d_col_ind,d_values, d_rows,d_columns, d_man_x,d_man_x_result,single_block_size );
        hipDeviceSynchronize(); 
        pt = d_man_x;
        d_man_x = d_man_x_result;
        d_man_x_result = pt;
    }
    gettimeofday(&t2, 0);

    double time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;

    //printf("Time to generate:  %3.11f ms \n", time);
  
    hipMemcpy(man_x, d_man_x, sizeof(double) * row_size, hipMemcpyDeviceToHost);

    
    if (print_what == 1){
        printDoubleArray(values,val_size);
        printArray(col_ind,val_size);
        printArray(row_ptr,row_size);
    }
    if (print_what == 2){
        printDoubleArray(man_x,row_size);
    }
    
    return 0;
}